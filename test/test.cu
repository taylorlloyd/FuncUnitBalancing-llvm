
#include <hip/hip_runtime.h>
__global__ void k(int *ret) {
  for(int i=0; i<100000; i++) {
    *ret = i;
  }
}

int main(int argc, char** argv) {
  int *d_argc;
  hipMalloc(&d_argc, sizeof(int));
  hipMemcpy(d_argc, &argc, sizeof(int), hipMemcpyHostToDevice);
  k<<<1,1>>>(d_argc);
  for(int i=0; i<100; i++) {
    argc += i;
  }
}
